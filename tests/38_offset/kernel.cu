
#include <hip/hip_runtime.h>
extern "C" __global__ void vecadd(int* A, int* B, int* C) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  C[id] = A[id] + B[id];
}
extern "C" __global__ void vecadd_with_offsets(int* A, int* B, int* C, size_t blockOff_x, size_t blockOff_y, size_t blockOff_z) {
  size_t id = (blockOff_x + blockIdx.x) * blockDim.x + threadIdx.x;
  //printf("block offset in vecadd : %i\n",blockOff_x);
  C[id] = A[id] + B[id];
}
extern "C" __global__ void blockadd(double* A, double* B, double* C) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  size_t j = blockIdx.y * blockDim.y + threadIdx.y;
  size_t SIZE = gridDim.y * blockDim.y;
  C[j * SIZE + i] = A[j * SIZE + i] + B[j * SIZE + i];
}
extern "C" __global__ void blockadd_with_offsets(double* A, double* B, double* C, size_t blockOff_x, size_t blockOff_y, size_t blockOff_z) {
  size_t i = (blockOff_x + blockIdx.x) * blockDim.x + threadIdx.x;
  size_t j = (blockOff_y + blockIdx.y) * blockDim.y + threadIdx.y;
  size_t SIZE = gridDim.y * blockDim.y;
  printf("using blockOff:(%lu, %lu), blockDim:(%i, %i), blockIdx:(%i, %i), threadIdx:(%i, %i), gridDim:(%i, %i) SIZE:%lu\n",blockOff_x, blockOff_y, blockDim.x, blockDim.y, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, gridDim.x, gridDim.y, SIZE);
  C[j * SIZE + i] = A[j * SIZE + i] + B[j * SIZE + i];
}
